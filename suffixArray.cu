#include "suffixArray.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sequence.h>
#include <thrust/sort.h>
#include <thrust/scan.h>
#include <thrust/execution_policy.h>
#include <thrust/for_each.h>

#include <iostream>
#include <memory>
#include <stdexcept>

using dstring = thrust::device_vector<char>;
using darray = thrust::device_vector<int>;

namespace
{
    constexpr unsigned int ALPHABET_SIZE = 128;
}

__host__
void suffixArray(const darray& s, int n, darray& out, darray& temp, 
            darray& s0, darray& s12, darray& rank, darray& rec_out) {
    std::cout << "N: " << n << std::endl;
    int n0 = (n + 2) / 3, n1 = (n + 1) / 3, n12 = 2*(n-1) / 3;

    thrust::sequence(temp.begin(), temp.begin() + n12);
    thrust::transform(temp.begin(), temp.begin() + n12, temp.begin(), [] __device__ (int x)
    {
        return 3*x/2 + 1;
    });

    {
        const auto data = s.data();
        for (int i=2; i>=0; --i)
        {
            thrust::stable_sort(temp.begin(), temp.begin() + n12, [i,data] __device__ (int a, int b)
            {
                return data[i+a] < data[i+b];
            });
        }
    }

    darray rec_in(n12 + 5);
    int cnt = 0;

    {
        darray temp_zero_one(n12 + 5);
        const auto temp_data = temp.data();
        const auto s_data = s.data();
        thrust::sequence(temp_zero_one.begin(), temp_zero_one.begin() + n12);
        thrust::transform(temp_zero_one.begin(), temp_zero_one.begin() + n12, temp_zero_one.begin(), [s_data, temp_data] __device__ (int position)
        {
            if (position == 0)
            {
                return 0;
            }
            for (int i=0; i<3; i++)
            {
                if (s_data[temp_data[position-1] + i] != s_data[temp_data[position] + i])
                {
                    return 1;
                }
            }
            return 0;
        });
        thrust::inclusive_scan(thrust::device, temp_zero_one.begin(), temp_zero_one.begin() + n12, temp_zero_one.begin());
        const auto burek = temp_zero_one.data();
        thrust::sequence(rec_in.begin(), rec_in.end());
        const auto rec_in_data = rec_in.data();
        thrust::for_each(thrust::device, temp_zero_one.begin(), temp_zero_one.begin() + n12, [burek, rec_in_data, temp_data, n1] __device__ (int i)
        {
            rec_in_data[temp_data[i] % 3 == 1 ? temp_data[i] / 3 : temp_data[i]/3 + n1 + 1] = burek[i] + 2; 
        });
        cnt = temp_zero_one[n12-1];
    }

    if (cnt != n12 + 1) {
        for (int i=0; i<3; i++)
        {
            rec_in[n12 + i + 1] = 0;
        }
        rec_in[n1] = 1;
        suffixArray(rec_in, n12+1, rec_out, temp, s0, s12, rank, rec_out);
        {
            auto burek = rec_out.data();
            thrust::sequence(s12.begin(), s12.begin() + n12);
            thrust::transform(s12.begin(), s12.begin() + n12, s12.begin(), [burek, n1] __device__ (int i)
            {
                ++i;
                return burek[i]<n1 ? 3*burek[i]+1 : 3*(burek[i]-n1)-1;
            });
        }

        {
            auto burek = s12.data();
            auto nobik = rank.data();
            thrust::sequence(temp.begin(), temp.begin() + n12);
            thrust::for_each(thrust::device, temp.begin(), temp.begin() + n12, [nobik, burek] __device__ (int i)
            {
                nobik[burek[i]] = i+1;
            });
            rank[n] = 0;
        }

        thrust::sequence(s0.begin(), s0.begin() + n0, 0, 3);

        {
            const auto data1 = rank.data();
            thrust::stable_sort(s0.begin(), s0.begin() + n0, [data1] __device__ (int a, int b)
            {
                return data1[a+1] < data1[b+1];
            });
            const auto data2 = s.data();
            thrust::stable_sort(s0.begin(), s0.begin() + n0, [data2] __device__ (int a, int b)
            {
                return data2[a] < data2[b];
            });
        }

        {
            const auto data_s = s.data();
            const auto data_r = rank.data();
            thrust::merge(thrust::device, s12.begin(), s12.begin()+n12, s0.begin(), s0.begin()+n0, out.begin(), [data_s, data_r] __device__ (int u, int v)
            {
                while (true)
                {
                    if (data_s[u] != data_s[v]) 
                    {
                        return data_s[u] < data_s[v];
                    }
                    if (u % 3 != 0 && v % 3 != 0)
                    {
                        return data_r[u] < data_r[v];
                    }
                    ++u;
                    ++v;
                }
            });
        }
    }

    for (int i = 0; i < 2*n/3; i++)
    {
        std::cout << temp[i] << ' ';
    }
    std::cout << std::endl;
}

__host__
void suffixArray(const std::string& in, int* out)
{
    const unsigned int n = in.size();
    
    if (n <= 1)
    {
        if (n == 1)
        {
            out[0] = 0;
        }
        return;
    }

    const unsigned int size = n + 3;

    darray s(in.data(), in.data() + in.size());
    s.resize(size);
    darray temp(size);
    darray s0(size);
    darray s12(size);
    darray rank(size);
    darray rec_out(size);
    darray device_out(n);

    suffixArray(s, n, device_out, temp, s0, s12, rank, rec_out);
}